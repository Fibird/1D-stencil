#include "hip/hip_runtime.h"

#include <iostream>
#include <algorithm>

using namespace std;

#define N          1024
#define RADIUS     3
#define BLOCK_SIZE 16

__global__ void stencil_1D(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + RADIUS;
	// Reads input elements into shared memory
	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS)
	{
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}
	// Synchronizes(ensure all the data is available)
	__syncthreads();
	// Applies the stencil
	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++)
		result += temp[lindex + offset];
	// Stores the result
	out[gindex] = result;
}


int main()
{
	int *in, *out;
	int *dev_in, *dev_out;

	// Allocs space for host copies and setup values
	in = (int*)malloc((N + 2 * RADIUS) * sizeof(int));
	fill_n(in, (N + 2 * RADIUS), 1);
	//fill_n(in, RADIUS, 0);
	//fill_n(in + N + RADIUS, RADIUS, 0);
	out = (int*)malloc((N + 2 * RADIUS) * sizeof(int));
	fill_n(out, (N + 2 * RADIUS), 1);
	//fill_n(out, RADIUS, 0);
	//fill_n(out + N + RADIUS, RADIUS, 0);

	// Alloc space for device copies
	hipMalloc((void**)&dev_in, (N + 2 * RADIUS) * sizeof(int));
	hipMalloc((void**)&dev_out, (N + 2 * RADIUS) * sizeof(int));

	// Copies to device
	hipMemcpy(dev_in, in, (N + 2 * RADIUS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_out, out, (N + 2 * RADIUS) * sizeof(int), hipMemcpyHostToDevice);

	// Launches stencil 1D kernel on GPU
	stencil_1D<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dev_in + RADIUS, dev_out + RADIUS);

	// Copies back to host
	hipMemcpy(out, dev_out, (N + 2 * RADIUS) * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = RADIUS; i < N; i++)
		cout << out[i] << " ";
	cout << endl;

	// cleanup
	free(in);
	free(out);
	hipFree(dev_in);
	hipFree(dev_out);
}

